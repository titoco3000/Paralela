#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//max de threads por bloco da gpu
#define MAX_THREADS 1024
//precisa ser impar
#define N_DIGITS 3000001

//extremamente ineficiente (mas é só pra debug, não usa na exec normal)
void printVec(int* vetor){
    int n;
    printf("[ ");
    for (int i = 0; i < N_DIGITS; i++)
    {
        hipMemcpy(&n, &vetor[i], sizeof(int), hipMemcpyDeviceToHost);
        printf("%d ",n);
    }
    puts("]");
}

__global__ void preencher_item(int *vetor, int offset){
    vetor[blockIdx.x*MAX_THREADS+threadIdx.x+offset] = 1;
}

void preencher(int *vetor){
    int blocos_completos = N_DIGITS/MAX_THREADS;
    int resto = N_DIGITS%MAX_THREADS;
    if(blocos_completos>0)
        preencher_item<<<blocos_completos,MAX_THREADS>>>(vetor, 0);
    if(resto>0)
        preencher_item<<<1,resto>>>(vetor, N_DIGITS-resto);
}

/*
Penteia os elementos do vetor no device.
Args:
- o offset a partir da qual vai pentear (contando do ultimo item até o primeiro + 1)
- o vetor no device
- o vetor de passagens (se tiver definido)
*/
__global__ void pentear(int offset,int *vetor_device) {
    //calcula indice a partir da threadID
    int i = N_DIGITS-((blockIdx.x*MAX_THREADS+threadIdx.x)*2 + offset+1);
    //carrega valor para proxima casa
    vetor_device[i - 1] = vetor_device[i - 1]*10 + vetor_device[i] / (i + 2);
    //deixa o resto no lugar
    vetor_device[i] %= (i + 2);
    //vetor_device[i]++;
}

/*
Penteia alguma qtd de casas a partir de um offset, contando a partir do ultimo.
As casas são alternadas: nunca penteia duas casas adjecentes
*/
void pentear_parte(int tamanho, int offset, int *vetor ){
    int blocos_completos = tamanho/MAX_THREADS;
    int resto = tamanho%MAX_THREADS;

    //printf("\ni: %d\tr: %d\toff: %d\n",blocos_completos,resto,offset);
    if(blocos_completos>0){
        pentear<<<blocos_completos,MAX_THREADS>>>(offset, vetor);
        //printVec(vetor);
    }
    if(resto>0){
        pentear<<<1,resto>>>( (tamanho-resto)*2+offset, vetor);
        //printVec(vetor);
    }
    
    // int feitos = 0;
    // int n = MAX_THREADS<tamanho-feitos?MAX_THREADS:tamanho-feitos;
    // pentear<<<1,n>>>(feitos+offset,vetor);
    // feitos+=n;
    // while (feitos<tamanho)
    // {
    //     n = MAX_THREADS<tamanho-feitos?MAX_THREADS:tamanho-feitos;
    //     pentear<<<1,n>>>((feitos*2)+offset,vetor);
    //     feitos+=n;
    // }
}

/*
Obtem o proximo digito a partir de um vetor totalmente penteado e armazena no arquivo
*/
void extrair(FILE* arquivo, int *vetor){
    int n;
    //obtem o valor da primeira casa do vetor do device
    hipMemcpy(&n, vetor, sizeof(int), hipMemcpyDeviceToHost);
    //determina quociente, que é o digito
    int q = n / 2;
    //determina resto, que vai substituir nessa casa
    int r = n % 2;
    n = r;
    //salva digito no arquivo
    fputc(q+'0',arquivo);
    
    //Se quiser imprimir na tela, basta descomentar essa linha
    //putchar(q+'0');
    
    //copia resto para vetor do device
    hipMemcpy(vetor,&n,sizeof(int), hipMemcpyHostToDevice);
}

/*
Prepara o vetor a cada inicio de passagem do pente
*/
void preparar_primeiro(int *vetor){
    int n;
    //obtem valor da ultima casa
    hipMemcpy(&n, &vetor[N_DIGITS-1], sizeof(int), hipMemcpyDeviceToHost);
    //multiplica por 10 (como nao tem item anterior q faça isso)
    n*=10;
    //copia informacao para device
    hipMemcpy(&vetor[N_DIGITS-1],&n,sizeof(int), hipMemcpyHostToDevice);
}

int main(){
    //abre arquivo de registro
    FILE *arquivo = fopen("out.txt","w+");
    if(arquivo){
        int *vetor;
        // Aloca memoria no device
        hipError_t malloc_result = hipMalloc((void**)&vetor, sizeof(int) * N_DIGITS);
        
        // Se teve sucesso
        if (malloc_result == hipSuccess)
        {
            fprintf(arquivo,"2.");
        
            // Preenche de 1s
            preencher(vetor);

            /*
            Penteia da ultima casa do vetor a penultima, depois da ultima a antepenultima, etc.
            até chegar numa seguinte a primeira
            */
            for (int i = 0; i < N_DIGITS/2; i++)
            {
                preparar_primeiro(vetor);
                //sao duas chamadas com offsets diferentes para pentear alternado, sem conflito
                pentear_parte(i+1,0,vetor);
                pentear_parte(i+1,1,vetor);
            }
            puts("Comecei a gerar digitos");
            //extrai primeiro digito menor digito depois da virgula de 'e' (deve ser 7)
            extrair(arquivo,vetor);    
            for (int _ = 0; _ <= N_DIGITS/2; _++)
            {
                preparar_primeiro(vetor);
                //sao duas chamadas com offsets diferentes para pentear alternado, sem conflito
                pentear_parte(N_DIGITS/2,0,vetor);
                pentear_parte(N_DIGITS/2,1,vetor);
                extrair(arquivo, vetor);
            }

            /*
            Penteia da ultima casa do vetor a segunda, depois da penultima a segunda, etc.
            até não ter nenhuma para pentear
            */
            for (int i = 0; i < N_DIGITS/2; i++)
            {
                //O offset que controla esse inicio
                pentear_parte(N_DIGITS/2-i-1, (i+1)*2,vetor);
                pentear_parte(N_DIGITS/2-i-1, (i+1)*2+1,vetor);
                extrair(arquivo, vetor);
            }
            // Libera memoria do device
            hipFree(vetor);
        }
        else{
            fprintf(stderr,"%s\n",hipGetErrorString(malloc_result));
        }       
    }
}